#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

//_global_ void ... -> device

double* readFromFile(int &rows);
void showMatrix(double *matrix, int rows);
void showFullMatrix(double *matrix, int rows);


double* gaussJordan(double *matrix, int rows)
{
	double *d_matrix = 0;
	double *ident_matrix = 0;
	int size_bytes = rows * rows * 2 * sizeof(matrix);		// number of bytes allocated on device mem
	dim3 threadsPerBlock(size_bytes, size_bytes);
	dim3 numBlocks((rows + size_bytes - 1) / size_bytes, (rows + size_bytes - 1) / size_bytes);


	/*hipMalloc(&d_matrix, size_bytes);
	hipMemcpy(d_matrix, matrix, size_bytes, hipMemcpyHostToDevice);*/

	ident_matrix = new double[rows*rows];

	for (int i = 0; i < rows; i++)
		for (int j = 0; j < rows; j++)
		{
			if (j == i)
				ident_matrix[i*rows + j] = 1;
			else
				ident_matrix[i*rows + j] = 0;
		}

	for (j = 0; j<rows; j++) {
		temp = j;

		/* finding maximum jth column element in last (rows-j) rows */

		for (i = j + 1; i<rows; i++)
			if (augmentedmatrix[i][j]>augmentedmatrix[temp][j])
				temp = i;

		if (fabs(augmentedmatrix[temp][j])<minvalue) {
			printf("\n Elements are too small to deal with !!!");
			break;
		}

		

		/* swapping row which has maximum jth column element */

		if (temp != j)
			for (k = 0; k<2 * rows; k++) {
				temporary = augmentedmatrix[j][k];
				augmentedmatrix[j][k] = augmentedmatrix[temp][k];
				augmentedmatrix[temp][k] = temporary;
			}

		/* performing row operations to form required identity matrix out of the input matrix */

		for (i = 0; i<rows; i++)
			if (i != j) {
				r = augmentedmatrix[i][j];
				for (k = 0; k<2 * rows; k++)
					augmentedmatrix[i][k] -= (augmentedmatrix[j][k] / augmentedmatrix[j][j])*r;
			}
			else {
				r = augmentedmatrix[i][j];
				for (k = 0; k<2 * rows; k++)
					augmentedmatrix[i][k] /= r;
			}

	}

	hipFree(d_matrix);
}

double* stickMatrix(double* matrix, double* ident_matrix, int rows)
{
	double* aug_matrix = new double[rows*rows * 2];
	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < rows; j++)
		{
			aug_matrix[i*rows*2 + j] = matrix[i*rows + j];
		}
		for (int j = rows; j < rows * 2; j++)
		{
			aug_matrix[i*rows*2 + j] = ident_matrix[i*rows + (j-rows)];
		}
	}
			
	return aug_matrix;
}

int main()
{
	double * matrix;
	CMeasure time;
	long long int time_table[3];
	int rows;

	matrix = readFromFile(rows);

	system("PAUSE");
	return 0;
}



//==========================================================================
//==========================================================================

double* readFromFile(int &rows)
{
	CFileStream file;
	double *matrix = 0;
	std::cout << "Insert file path\n";
	file.openFile();
	matrix = file.readData(matrix, rows);		// !!! size * 2 in allocation
	return matrix;
}

void showMatrix(double *matrix, int rows)
{
	if (rows > 0)
	{
		for (int i = 0; i < rows; i++)
		{
			for (int j = 0; j < rows; j++)
			{
				std::cout << matrix[i*rows + j] << " ";
			}
			std::cout << std::endl;
		}
	}
}

void showFullMatrix(double *matrix, int rows)
{
	if (rows > 0)
	{
		for (int i = 0; i < rows; i++)
		{
			for (int j = 0; j < 2*rows; j++)
			{
				std::cout << matrix[i*2*rows + j] << " ";
			}
			std::cout << std::endl;
		}
	}
}


//class CMatrix
//{
//	CMatrix(int size) : size(size)
//	{
//		matrix = new double[size];
//	}
//	~CMatrix()
//	{
//		delete[] matrix;
//	}
//
//	CMatrix operator+(const CMatrix &augmented)
//	{
//		CMatrix augmentedMatrix(size * 2);
//		// sssssssss
//	}
//
//	double *matrix;
//
//private:
//
//	int size;
//
//};
