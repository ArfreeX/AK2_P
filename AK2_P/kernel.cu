#include "stdafx.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

// _global_ void ... -> device
 


int** readFromFile(int &rows)
{
	CFileStream file;
	int **matrix = 0;
	std::cout << "Insert file path\n";
	file.openFile();
	matrix = file.readData(matrix, rows);
	return matrix;
}

void showMatrix(int **matrix, int rows)
{
	if (rows > 0)
	{
		for (int i = 0; i < rows; i++)
		{
			for (int j = 0; j < rows; j++)
				std::cout << matrix[i][j] << " ";
			std::cout << std::endl;
		}
		
	}
}
int main()
{
	int **matrix = 0, **d_matrix = 0, rows; // todo: delete []

	matrix = readFromFile(rows);
	showMatrix(matrix, rows);
	
	int size_bytes = rows * rows * 4;		// number of bytes allocated on device mem

	// hipMalloc((void**)&d_matrix, sizeBytes); 

	system("PAUSE");
	return 0;
}