#include "stdafx.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

//_global_ void ... -> device

double* readFromFile(int &rows)
{
	CFileStream file;
	double *matrix = 0;
	std::cout << "Insert file path\n";
	file.openFile();
	matrix = file.readData(matrix, rows);
	return matrix;
}

void showMatrix(double *matrix, int rows)
{
	if (rows > 0)
	{
		for (int i = 0; i < rows; i++)
		{
			for (int j = 0; j < rows; j++)
			{
				std::cout << matrix[i*rows+j] << " ";
			}
			std::cout << std::endl;
		}
	}
}
int main()
{
	CMeasure time;
	long long int time_table[3];

	double *matrix = 0, *d_matrix = 0;
	int rows;
	double *ident_matrix;


	matrix = readFromFile(rows);
	
	ident_matrix = new double[rows*rows];

	for(int i = 0 ; i < rows; i++)
		for (int j = 0; j < rows; j++)
		{
			if (j == i)
				ident_matrix[i*rows + j] = 1;
			else
				ident_matrix[i*rows + j] = 0;
		}


	showMatrix(matrix, rows);
	
	int size_bytes = rows * rows * 4;		// number of bytes allocated on device mem
	dim3 threadsPerBlock(size_bytes, size_bytes);
	dim3 numBlocks((rows + size_bytes - 1) / size_bytes, (rows + size_bytes - 1) / size_bytes);

	hipMalloc(&d_matrix, size_bytes); 
	hipMemcpy(d_matrix, matrix, size_bytes, hipMemcpyHostToDevice);

	time.start();
	// TODO:
	time_table[0] = time.elapsed();

	hipMemcpy(matrix, d_matrix, size_bytes, hipMemcpyDeviceToHost);

	showMatrix(matrix, rows);				// results

	hipFree(d_matrix);
	delete[] matrix;
	delete[] ident_matrix;
	system("PAUSE");
	return 0;
}
