#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#define MIN_ELEMENT 0.000005
#define BLOCK_SIZE 512 // needs to be checked for proper values
#define MAX_ELEMENT 15

__global__ void swap(double* matrix, int rows, int temp,int j)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x < rows * 2)
	{
		double temporary = matrix[j*rows * 2 + x];
		matrix[j*rows * 2 + x] = matrix[temp*rows * 2 + x];
		matrix[temp*rows * 2 + x] = temporary;
	}
}

__global__ void gjAlgorithm(double* matrix, int rows, double temp, int i,int j)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (x < rows * 2)
	{
		matrix[i*rows * 2 + x] -= (matrix[j*rows * 2 + x] / matrix[j*rows * 2 + j])*temp;
	}
}

__global__ void gjAlgorithm2(double* matrix, int rows, double temp, int i, int j )
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x < rows * 2)
	{
		matrix[i*rows * 2 + x] /= temp;
	}
}

void randomMatrixToFile(int rows)
{
	CFileStream file;
	const int SEED = int(time(NULL));
	srand(SEED);
	double *matrix = new double[rows*rows];

	for (int i = 0; i < rows*rows; ++i)
	{	
		matrix[i] = rand() % MAX_ELEMENT + 1;
	}

	file.write(matrix, rows);
	delete[] matrix;
}

double* readFromFile(int &rows);
void showMatrix(double *matrix, int rows);
void showFullMatrix(double *matrix, int rows);
double* stickMatrix(double* matrix, double* ident_matrix, int rows);

double* gaussJordan(double *matrix, int rows)
{
	int temp;
	double *d_matrix = 0;
	double *ident_matrix = 0;
	int size_bytes = rows * rows * 2 * sizeof(double);		// number of bytes allocated on device mem
	dim3 threadsPerBlock(BLOCK_SIZE);
	dim3 numBlocks((2*rows + BLOCK_SIZE - 1) / BLOCK_SIZE);

	auto err = hipMalloc(&d_matrix, size_bytes);
	if (err != hipSuccess) { std::cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << std::endl; }

	ident_matrix = new double[rows*rows];
	for (int i = 0; i < rows; i++) 
	{
		for (int j = 0; j < rows; j++)
		{
			if (j == i)
				ident_matrix[i*rows + j] = 1;
			else
				ident_matrix[i*rows + j] = 0;
		}
	}

	double *augmentedmatrix = stickMatrix(matrix, ident_matrix, rows);


//=================================================================================================================================================
//															Obliczanie rownolegle - petla GJ
//=================================================================================================================================================

	for (int j = 0; j<rows; j++) 
	{
		temp = j;

		/* finding maximum jth column element in last (rows-j) rows */
		for (int i = j + 1; i < rows; i++) {
			if (augmentedmatrix[i*rows * 2 + j] > augmentedmatrix[temp*rows * 2 + j])
				temp = i;
		}
		
		if (fabs(augmentedmatrix[temp*rows*2 + j])<MIN_ELEMENT) {
			printf("\n Elements are too small to deal with !!!\n");
			break;
		}

		/* swapping row which has maximum jth column element */
		//KERNEL?
		double temporary;
		if (temp != j)
		{
			err = hipMemcpy(d_matrix, augmentedmatrix, size_bytes, hipMemcpyHostToDevice);
			if (err != hipSuccess) { std::cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << std::endl; }

			swap << < numBlocks, threadsPerBlock >> > (d_matrix, rows, temp, j);

			err = hipMemcpy(augmentedmatrix, d_matrix, size_bytes, hipMemcpyDeviceToHost);
			if (err != hipSuccess) { std::cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << std::endl; }
			
		}
	
		
		/* performing row operations to form required identity matrix out of the input matrix */
		//KERNEL?
		double r;
		for (int i = 0; i < rows; i++)
		{
			r = augmentedmatrix[i*rows * 2 + j];
			if (i != j)
			{
				err = hipMemcpy(d_matrix, augmentedmatrix, size_bytes, hipMemcpyHostToDevice);
				if (err != hipSuccess) { std::cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << std::endl; }

				gjAlgorithm << < numBlocks, threadsPerBlock >> > (d_matrix, rows, r, i, j);

				err = hipMemcpy(augmentedmatrix, d_matrix, size_bytes, hipMemcpyDeviceToHost);
				if (err != hipSuccess) { std::cout << hipGetErrorString(err) 
					<< " in " << __FILE__ << " at line " << __LINE__ << std::endl; }
			}
				
			else
			{
				err = hipMemcpy(d_matrix, augmentedmatrix, size_bytes, hipMemcpyHostToDevice);
				if (err != hipSuccess) { std::cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << std::endl; }
				
				gjAlgorithm2 << < numBlocks, threadsPerBlock >> > (d_matrix, rows, r, i, j);

				err = hipMemcpy(augmentedmatrix, d_matrix, size_bytes, hipMemcpyDeviceToHost);
				if (err != hipSuccess) { std::cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << std::endl; }
			}
		}
	}

	hipFree(d_matrix);

	return augmentedmatrix;
}

double* stickMatrix(double* matrix, double* ident_matrix, int rows)
{
	double* aug_matrix = new double[rows*rows * 2];
	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < rows; j++)
		{
			aug_matrix[i*rows*2 + j] = matrix[i*rows + j];
		}
		for (int j = rows; j < rows * 2; j++)
		{
			aug_matrix[i*rows*2 + j] = ident_matrix[i*rows + (j-rows)];
		}
	}
			
	return aug_matrix;
}

int main()
{
	double * matrix;
	CMeasure time;
	long long int time_table[3];
	int rows = 500;
	randomMatrixToFile(rows);
	matrix = readFromFile(rows);
	//showMatrix(matrix, rows);
	std::cout << "\n\n";
	showFullMatrix(gaussJordan(matrix, rows), rows);
	std::cout << "\n\n\n";
	return 0;
}



//==========================================================================
//==========================================================================

double* readFromFile(int &rows)
{
	CFileStream file;
	double *matrix = 0;
	std::cout << "Insert file path\n";
	file.openFile();
	matrix = file.readData(matrix, rows);		// !!! size * 2 in allocation
	return matrix;
}

void showMatrix(double *matrix, int rows)
{
	if (rows > 0)
	{
		for (int i = 0; i < rows; i++)
		{
			for (int j = 0; j < rows; j++)
			{
				std::cout << matrix[i*rows + j] << " ";
			}
			std::cout << std::endl;
		}
	}
}

void showFullMatrix(double *matrix, int rows)
{
	if (rows > 0)
	{
		for (int i = 0; i < rows; i++)
		{
			for (int j = 0; j < 2*rows; j++)
			{
				std::cout << matrix[i*2*rows + j] << " ";
			}
			std::cout << std::endl;
		}
	}
}


//class CMatrix
//{
//	CMatrix(int size) : size(size)
//	{
//		matrix = new double[size];
//	}
//	~CMatrix()
//	{
//		delete[] matrix;
//	}
//
//	CMatrix operator+(const CMatrix &augmented)
//	{
//		CMatrix augmentedMatrix(size * 2);
//		// sssssssss
//	}
//
//	double *matrix;
//
//private:
//
//	int size;
//
//};
